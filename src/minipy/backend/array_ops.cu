#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <stdexcept>

#include "array_ops.hpp"

namespace minipy {
__global__ void add_kernel(const double *a, const double *b, double *result,
                           size_t size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    result[idx] = a[idx] + b[idx];
  }
}

std::vector<double> ArrayOps::add_gpu(const std::vector<double> &a,
                                      const std::vector<double> &b) {
  if (a.size() != b.size()) {
    throw std::invalid_argument("Arrays must have the same size");
  }

  size_t size = a.size();
  std::vector<double> result(size);

  double *d_a, *d_b, *d_result;
  hipMalloc(&d_a, size * sizeof(double));
  hipMalloc(&d_b, size * sizeof(double));
  hipMalloc(&d_result, size * sizeof(double));

  hipMemcpy(d_a, a.data(), size * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b.data(), size * sizeof(double), hipMemcpyHostToDevice);

  int threadsPerBlock = 256;
  int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

  add_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_result, size);

  hipMemcpy(result.data(), d_result, size * sizeof(double),
             hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_result);

  return result;
}

__global__ void dot_kernel(const double *a, const double *b, double *result,
                           size_t size) {
  __shared__ double shared_mem[256];
  int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  double sum = 0.0;
  if (idx < size) {
    sum = a[idx] * b[idx];
  }

  shared_mem[tid] = sum;
  __syncthreads();

  for (int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      shared_mem[tid] += shared_mem[tid + s];
    }
    __syncthreads();
  }

  if (tid == 0) {
    atomicAdd(result, shared_mem[0]);
  }
}

std::vector<double> ArrayOps::dot_gpu(const std::vector<double> &a,
                                      const std::vector<double> &b) {
  if (a.size() != b.size()) {
    throw std::invalid_argument("Arrays must have the same size");
  }

  size_t size = a.size();
  double *d_a, *d_b, *d_result;
  hipMalloc(&d_a, size * sizeof(double));
  hipMalloc(&d_b, size * sizeof(double));
  hipMalloc(&d_result, sizeof(double));

  hipMemcpy(d_a, a.data(), size * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b.data(), size * sizeof(double), hipMemcpyHostToDevice);
  hipMemset(d_result, 0, sizeof(double));

  int threadsPerBlock = 256;
  int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

  dot_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_result, size);

  double host_result;
  hipMemcpy(&host_result, d_result, sizeof(double), hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_result);

  return std::vector<double>{host_result};
}
}  // namespace minipy
