#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <stdexcept>

#include "array_ops.hpp"

namespace minipy {
__global__ void add_kernel(const double *a, const double *b, double *result,
                           size_t size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    result[idx] = a[idx] + b[idx];
  }
}

std::vector<double> ArrayOps::add_gpu(const std::vector<double> &a,
                                      const std::vector<double> &b) {
  if (a.size() != b.size()) {
    throw std::invalid_argument("Arrays must have the same size");
  }

  size_t size = a.size();
  std::vector<double> result(size);

  double *d_a, *d_b, *d_result;
  hipMalloc(&d_a, size * sizeof(double));
  hipMalloc(&d_b, size * sizeof(double));
  hipMalloc(&d_result, size * sizeof(double));

  hipMemcpy(d_a, a.data(), size * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b.data(), size * sizeof(double), hipMemcpyHostToDevice);

  int threadsPerBlock = 256;
  int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

  add_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_result, size);

  hipMemcpy(result.data(), d_result, size * sizeof(double),
             hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_result);

  return result;
}
}  // namespace minipy
